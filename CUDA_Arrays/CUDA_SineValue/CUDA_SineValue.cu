#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void sineof(float *X, float *Y){
	int idx = blockIdx.x;
	Y[idx] = sinf(X[idx]);
}

int main(){
	float *X,*Y, N; //program vars
	float *d_x, *d_y; //device vars
	int size = sizeof(float);

	printf("Enter number of elements: ");
	scanf("%f",&N);

	X = (float*)malloc(sizeof(float)*N);
	Y = (float*)malloc(sizeof(float)*N);

	printf("Enter elements in rad:\n");
	for(int i=0; i<N; i++){
		scanf("%f ",&X[i]);
	}

	//Allocate space for device copies of a,b,c
	hipMalloc((void**)&d_x,size*N);
	hipMalloc((void**)&d_y,size*N);

	//setup input values
	hipMemcpy(d_x,X,size*N,hipMemcpyHostToDevice);
	hipMemcpy(d_y,Y,size*N,hipMemcpyHostToDevice);

	//launch add kernel on GPU
	sineof<<<N,1>>>(d_x,d_y);

	//copy result back to host
	hipMemcpy(Y,d_y,sizeof(float)*N,hipMemcpyDeviceToHost);
	printf("Result:\n");
	for(int i=0; i<N; i++){
		printf("Y%d = %f \n",i,Y[i]);
	}

	//Cleanup
	hipFree(d_x);
	hipFree(d_y);
	return 0;

}
