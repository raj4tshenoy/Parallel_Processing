#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void add1(int *a, int *b, int *c){
	int idx = blockIdx.x;
	c[idx] = a[idx] + b[idx];
}

__global__ void add2(int* a, int* b, int* c){
	int idx = threadIdx.x;
	c[idx] = a[idx] + b[idx];
}

__global__ void add3(int* a, int* b, int* c){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	c[idx] = a[idx] + b[idx];
}

int main(){
	int N; //number of elements
	int *d_a, *d_b, *d_c; //Device copies
	int size = sizeof(int); //size var
	int *A, *B, *C; //vectors

	printf("Enter number of elements: ");
	scanf("%d",&N);

	A = (int*)malloc(sizeof(int)*N);
	B = (int*)malloc(sizeof(int)*N);
	C = (int*)malloc(sizeof(int)*N);

	printf("Enter elements a <space> b:\n");
	for(int i=0; i<N; i++){
		scanf("%d %d",&A[i],&B[i]);
	}

	//Allocate space for device copies of a,b,c
	hipMalloc((void**)&d_a,size*N);
	hipMalloc((void**)&d_b,size*N);
	hipMalloc((void**)&d_c,size*N);

	//setup input values
	hipMemcpy(d_a,A,size*N,hipMemcpyHostToDevice);
	hipMemcpy(d_b,B,size*N,hipMemcpyHostToDevice);

	printf("\n// LAUNCHING ADD1 //\n");
	//launch add kernel on GPU
	dim3 Blocks1(N,1,1);
	dim3 Threads1(1,1,1);
	add1<<<Blocks1,Threads1>>>(d_a,d_b,d_c);

	//copy result back to host
	hipMemcpy(C,d_c,size*N,hipMemcpyDeviceToHost);
	printf("Result:\n");

	for(int i=0; i<N; i++)
		printf("%d ",C[i]);



	printf("\n// LAUNCHING ADD2 //\n");
	//launch add kernel on GPU
	dim3 Blocks2(1,1,1);
	dim3 Threads2(N,1,1);
	add1<<<Blocks2,Threads2>>>(d_a,d_b,d_c);

	//copy result back to host
	hipMemcpy(C,d_c,size*N,hipMemcpyDeviceToHost);
	printf("Result:\n");

	for(int i=0; i<N; i++)
		printf("%d ",C[i]);


	printf("\n// LAUNCHING ADD3 //\n");
	//launch add kernel on GPU
	dim3 Blocks3(ceilf(N/256),1,1);
	dim3 Threads3(256,1,1);
	add1<<<Blocks3,Threads3>>>(d_a,d_b,d_c);

	//copy result back to host
	hipMemcpy(C,d_c,size*N,hipMemcpyDeviceToHost);
	printf("Result:\n");

	for(int i=0; i<N; i++)
		printf("%d ",C[i]);


	//Cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;

}
