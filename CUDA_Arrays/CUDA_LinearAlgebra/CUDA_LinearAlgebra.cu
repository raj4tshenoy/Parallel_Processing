#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *X, int *Y, int *alpha){
	int idx = blockIdx.x;
	Y[idx] = ((*alpha)*(X[idx])) + Y[idx];
}

int main(){
	int alpha,*X,*Y, N; //program vars
	int *d_x, *d_y, *d_a; //device vars
	int size = sizeof(int);

	printf("Enter number of elements and alpha: ");
	scanf("%d %d",&N, &alpha);

	X = (int*)malloc(sizeof(int)*N);
	Y = (int*)malloc(sizeof(int)*N);

	printf("Enter elements x <space> y:\n");
	for(int i=0; i<N; i++){
		scanf("%d %d",&X[i],&Y[i]);
	}

	//Allocate space for device copies of a,b,c
	hipMalloc((void**)&d_x,size*N);
	hipMalloc((void**)&d_y,size*N);
	hipMalloc((void**)&d_a,size);

	//setup input values
	hipMemcpy(d_a,&alpha,size,hipMemcpyHostToDevice);
	hipMemcpy(d_x,X,size*N,hipMemcpyHostToDevice);
	hipMemcpy(d_y,Y,size*N,hipMemcpyHostToDevice);

	//launch add kernel on GPU
	add<<<N,1>>>(d_x,d_y,d_a);

	//copy result back to host
	hipMemcpy(Y,d_y,size*N,hipMemcpyDeviceToHost);
	printf("Result:\n");
	for(int i=0; i<N; i++){
		printf("Y%d = %d \n",i,Y[i]);
	}

	//Cleanup
	hipFree(d_a);
	hipFree(d_x);
	hipFree(d_y);
	return 0;

}
