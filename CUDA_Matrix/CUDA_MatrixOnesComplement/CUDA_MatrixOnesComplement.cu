#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void onesc(int *A, int *B){
	int ele = threadIdx.x, row=blockIdx.x, no_eles = blockDim.x, no_rows = gridDim.x;
	if(row!=0 && ele!=0 && row!=no_rows-1 && ele != no_eles-1){
		int cur = A[row*no_eles + ele];
		int binaryNum[32], bin=0;
		int i = 0;
		while (cur > 0) { //find binary equivalent
			binaryNum[i] = cur % 2;
		    cur = cur / 2;
		    i++;
		}

		for(int k=0;k<i;k++){ //find ones complement
			if(binaryNum[k]==1){
				binaryNum[k] = 0;
			}
			else{
				binaryNum[k] = 1;
			}
		}

		for (int j = i - 1; j >= 0; j--) //put to integer element
		    bin = bin*10 + binaryNum[j];

		B[row*no_eles + ele] = bin;
	}
	else{
		B[row*no_eles + ele] = A[row*no_eles + ele];
	}
}

int main(){
    int *a, *t, m, n;
    int *d_a, *d_t;
    printf("Enter the value of m: "); scanf("%d",&m);
    printf("Enter the value of n: "); scanf("%d",&n);
    int size = sizeof(int)*m*n;
    a=(int*)malloc(size);
    t=(int*)malloc(size);
    printf("Enter input matrix: \n");
    for(int i=0; i<m*n; i++)
        scanf("%d",&a[i]);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_t,size);

    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);

    onesc<<<m,n>>>(d_a,d_t);

    hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
    printf("Resultant matrix:\n");
    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            printf("%d ",t[i*n+j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_t);
    return 0;
}
