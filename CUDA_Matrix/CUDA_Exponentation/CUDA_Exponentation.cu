/*
 *
 *	PROBLEM STATEMENT:
 *
 *	Each row of the given matrix is exponentiated as many times as the row number. parallely
 *	eg. row 1 is the same. row 2 is squared. row 3 is cubed and so on...
 *
 *
 */

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void exponentiate(int *A, int *B){
	int ele = threadIdx.x, row=blockIdx.x, no_eles = blockDim.x;
	B[row*no_eles + ele] = 1;
	for(int i=0; i<=row; i++){
		B[row*no_eles + ele] = B[row*no_eles + ele] * A[row*no_eles + ele];
	}
}

int main(){
    int *a, *t, m, n;
    int *d_a, *d_t;
    printf("Enter the value of m: "); scanf("%d",&m);
    printf("Enter the value of n: "); scanf("%d",&n);
    int size = sizeof(int)*m*n;
    a=(int*)malloc(size);
    t=(int*)malloc(size);
    printf("Enter input matrix: \n");
    for(int i=0; i<m*n; i++)
        scanf("%d",&a[i]);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_t,size);

    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);

    exponentiate<<<m,n>>>(d_a,d_t);

    hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
    printf("Resultant matrix:\n");
    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            printf("%d ",t[i*n+j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_t);
    return 0;
}
