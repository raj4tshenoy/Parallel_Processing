#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void add1(int *A, int *B, int *C, int n){ //each thread computes the sum of elements row-wise
	int row = threadIdx.x;
	for(int i=0;i<n;i++){
		C[row*n+i] = A[row*n +i] + B[row*n+i];
	}

}

__global__ void add2(int *A, int *B, int *C, int m){ //each thread computes the sum of elements column-wise
	int col = threadIdx.x;
	for(int i=0;i<m;i++){
		C[col*m+i] = A[col*m +i] + B[col*m+i];
	}

}

__global__ void add3(int *A, int *B, int *C){ //each thread computes sum of 2 elements
	int ele = threadIdx.x, row=blockIdx.x, no_eles = blockDim.x;
	C[row*no_eles + ele] = A[row*no_eles + ele] + B[row*no_eles + ele];
}

int main(){
    int *a, *b, *t, m, n;
    int *d_a, *d_b, *d_t;
    printf("Enter the value of m: "); scanf("%d",&m);
    printf("Enter the value of n: "); scanf("%d",&n);
    int size = sizeof(int)*m*n;
    a=(int*)malloc(size);
    b=(int*)malloc(size);
    t=(int*)malloc(size);
    printf("Enter input matrix A: \n");
    for(int i=0; i<m*n; i++)
        scanf("%d",&a[i]);

    printf("Enter input matrix B: \n");
        for(int i=0; i<m*n; i++)
            scanf("%d",&b[i]);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_t,size);

    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

    add1<<<1,m>>>(d_a,d_b,d_t,n);

        hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
        printf("Resultant matrix ADD3:\n");
        for(int i=0; i<m; i++){
            for(int j=0; j<n; j++){
                printf("%d ",t[i*n+j]);
            }
            printf("\n");
     }

    add2<<<1,n>>>(d_a,d_b,d_t,m);

    hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
    printf("Resultant matrix ADD3:\n");
    for(int i=0; i<m; i++){
    	for(int j=0; j<n; j++){
    		printf("%d ",t[i*n+j]);
        }
    	printf("\n");
    }

    add3<<<m,n>>>(d_a,d_b,d_t);

    hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
    printf("Resultant matrix ADD3:\n");
    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            printf("%d ",t[i*n+j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_t);
    return 0;
}
